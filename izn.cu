#include "hip/hip_runtime.h"
#include "IZN.h"
#include "HADES.h"
#include <hip/hip_runtime.h>

#define CUDA_MAX_NUMBER_OF_THREADS_PER_BLOCK 1024


// This has been CUDAfied - hence why we are not using references & now use pointers.
__global__ void izhikevich_neuronCUDA(const iznParameters *params, iznState *stateArray, float delT, float *I_in_array)
{
	float vDot, uDot;
	float v_temp;
#ifdef __NVCC__
    int neuronThreadIdx = (blockIdx.x * blockDim.x) + threadIdx.x;
    iznState *state = &stateArray[neuronThreadIdx];
    float *I_in = &I_in_array[neuronThreadIdx];
    if(neuronThreadIdx >= nNeurons)
    {
        return; // do nothing if our index is greater than the number of neurons we have in the array.
    }
#endif // __NVCC__
#ifdef __VITIS_HLS__
//	#pragma HLS inline
	#pragma HLS dataflow
//	#pragma HLS pipeline
#endif // __VITIS_HLS__
	
	if (state->v >= params->rstV) {
		state->v		= params->c;
		state->u		= state->u + params->d;
		state->spike = 1;
	}
	else {
		vDot		= (0.04f * (state->v) * (state->v)) + (5 * (state->v)) + 140.0f - state->u + *I_in; // can run in parallel with line 170
		uDot		= (params->a * ((params->b * state->v) - state->u)); // can run in parallel with 169

#ifdef __VITIS_HLS__
	#pragma HLS pipeline II=1
#endif // __VITIS_HLS__
#ifndef __NVCC__
        state->u		= HADES_euler(delT, state->u, uDot); // can run in parallel with 177 and 178 (can be executed as serial step #1)
		v_temp		    = HADES_euler(delT, state->v, vDot); // serial step #1
#else // __NVCC__
        state->u        = (state->u + (delT * uDot));
        v_temp          = (state->v + (delT * vDot));
#endif // __NVCC__
		state->v		= (v_temp >= params->rstV) ? params->rstV : v_temp; // serial step #2
		state->spike = 0;
	}
}

__host__ void izhikevich_initStates(iznState *neuronStates)
{
#ifdef __VITIS_HLS__
	#pragma HLS dataflow
	#pragma HLS pipeline
#endif // __VITIS_HLS__
	
iznInitLoop: 
	for (int i = 0; i < nNeurons; i++) {

#ifdef __VITIS_HLS__
	#pragma HLS dataflow
	#pragma HLS pipeline
	#pragma HLS unroll
#endif // __VITIS_HLS__

		neuronStates[i].v		= -70.0f;
		neuronStates[i].u		= neuronStates[i].v * (float)IZN_b;
		neuronStates[i].spike	= (spikePulse)0;
	}
}

__host__ void izhikevich_population(resetMemory *rst, float *delT, float I_in[nNeurons], iznState outputStates[nNeurons], iznState outputStatesGPU[nNeurons])
{
	// Memory items
	static iznState iznStates[nNeurons];
	static const iznParameters params = iznParameters_default;

#ifdef __NVCC__
    static iznState *d_iznStates;
    static iznParameters *d_params;
    static float *d_I_in;
    static iznState *d_outputStates;

    int numThreadsPerBlock = CUDA_MAX_NUMBER_OF_THREADS_PER_BLOCK;
    int numBlocks = (nNeurons / CUDA_MAX_NUMBER_OF_THREADS_PER_BLOCK) + 1;
    //delT and params Can be placed into shared memory.

#endif // __NVCC__

#ifdef __VITIS_HLS__
	#pragma HLS dataflow
	#pragma HLS pipeline
#endif // __VITIS_HLS__
	
	// Neuron state initialization
	if (*rst == (resetMemory)1) {
		izhikevich_initStates(iznStates);
#ifdef __NVCC__
        hipMalloc((void**) &d_iznStates,    sizeof(iznState) * nNeurons);
        hipMalloc((void**) &d_params,       sizeof(iznParameters));
        hipMalloc((void**) &d_I_in,         sizeof(float) * nNeurons);
        hipMalloc((void**) &d_outputStates, sizeof(iznState) * nNeurons);

        hipMemcpy(d_iznStates, iznStates,   sizeof(iznState) * nNeurons, hipMemcpyHostToDevice);
        hipMemcpy((void *) d_params,        (const void *) &params,      sizeof(iznParameters),       hipMemcpyHostToDevice);
        hipMemcpy(d_I_in,      I_in,        sizeof(float) * nNeurons,    hipMemcpyHostToDevice);
        hipMemset(d_outputStates, 0x00,     sizeof(iznState) * nNeurons);
#endif // __NVCC__
	}
	else {
	iznPopLoop: 
#ifndef __NVCC__
		for (int i = 0; i < nNeurons; i++) {
			izhikevich_neuron( params, iznStates[i], *delT, I_in[i]); // CPU Run
		}
#endif // __NVCC__
#ifdef __NVCC__
        izhikevich_neuronCUDA<<<numBlocks, numThreadsPerBlock>>>(d_params, d_iznStates, *delT, d_I_in); // GPU run
#endif // __NVCC__	
    }
	
iznOutputLoop: 
#ifndef __NVCC__
	for (int i = 0; i < nNeurons; i++) {

#ifdef __VITIS_HLS__
	#pragma HLS dataflow
	#pragma HLS unroll factor = 50
	#pragma HLS pipeline
#endif // __VITIS_HLS__

		outputStates[i] = iznStates[i];
	}
#endif // __NVCC__

    // now copy the results back from the device, and free memory.
    hipMemcpy((void *)outputStatesGPU, (void *) d_outputStates, sizeof(iznState) * nNeurons, hipMemcpyDeviceToHost);
    //hipFree(d_iznStates);
    //hipFree(d_params);
    //hipFree(d_I_in);
    //hipFree(d_outputStates);
}


